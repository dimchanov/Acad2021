// UCSC CMPE220 Advanced Parallel Processing
// Prof. Heiner Leitz
// Author: Marcelo Siero.
// Modified from code by:: Andreas Goetz (agoetz@sdsc.edu)
// CUDA program to perform 1D stencil operation in parallel on the GPU
//
// /* FIXME */ COMMENTS ThAT REQUIRE ATTENTION


#include <hip/hip_runtime.h>
#include <cuda_device_runtime_api.h>
#include <iostream>
#include <stdio.h>

// define vector length, stencil radius,

#define RADIUS 5
#define BLOCKSIZE 1024

int gridSize;
int blockSize = BLOCKSIZE;
float time_milliseconds = 0;
int N;

void cudaErrorCheck()
{
    std::cout << "====================================================="
              << std::endl;
    hipError_t error = hipGetLastError();
    std::string errorName = std::string(hipGetErrorName(error));
    std::cout << "Error name: " << errorName << std::endl;

    std::string errorDescription = std::string(hipGetErrorString(error));
    std::cout << "**** " << errorDescription << " ****" << std::endl;
    std::cout << "====================================================="
              << std::endl;
}

void start_timer(hipEvent_t* start)
{
    hipEventCreate(start);
    hipEventRecord(*start);
}

float stop_timer(hipEvent_t* start, hipEvent_t* stop)
{
    hipEventCreate(stop);
    hipEventRecord(*stop);

    hipEventSynchronize(*stop);
    hipEventElapsedTime(&time_milliseconds, *start, *stop);
    return (time_milliseconds);
}

hipDeviceProp_t prop;
int device;
void getDeviceProperties()
{
    hipGetDevice(&device);
    hipGetDeviceProperties(&prop, device);
    std::cout << "Major and minor cuda capabilities are: " << prop.major << ", "
              << prop.minor << std::endl;
    std::cout << "Total device global memory is : "
              << static_cast<int>(prop.totalGlobalMem) << " bytes" << std::endl;
    std::cout << "Size of shared memory per block is : "
              << static_cast<int>(prop.sharedMemPerBlock) << " bytes"
              << std::endl;
    std::cout << "Number of registers per block is: " << prop.regsPerBlock
              << std::endl;
    std::cout << "Warp size is : " << prop.warpSize << " threads" << std::endl;
    std::cout << "Max number of threads per block is : "
              << prop.maxThreadsPerBlock << std::endl;
    std::cout << "Number of multiprocessors is : " << prop.multiProcessorCount
              << " per device" << std::endl;
    std::cout << "Number of Maximum number of threads per block dimension "
                 "(x,y,z) per device: "
              << prop.maxThreadsDim[0] << ", " << prop.maxThreadsDim[1] << ", "
              << prop.maxThreadsDim[2] << std::endl;
    std::cout << "Maximumum number of blocks per grid dimension "
              << prop.maxGridSize[0] << ", " << prop.maxGridSize[0] << ", "
              << prop.maxGridSize[1] << ", " << prop.maxGridSize[2] << std::endl;
}

void newline() { std::cout << std::endl; };

void printThreadSizes()
{
    int noOfThreads = gridSize * blockSize;
    printf("Blocks            = %d\n", gridSize); // no. of blocks to launch.
    printf("Threads per block = %d\n", blockSize); // no. of threads to launch.
    printf("Total threads     = %d\n", noOfThreads);
    printf("Number of grids   = %d\n", (N + noOfThreads - 1) / noOfThreads);
}

// -------------------------------------------------------
// CUDA device function that performs 1D stencil operation
// -------------------------------------------------------

__global__ void stencil_1D (double *in, double *out, double dim) {

    long gindex = threadIdx.x + blockDim.x * blockIdx.x;
    int stride = gridDim.x * blockDim.x;

  // Go through all data
  // Step all threads in a block to avoid synchronization problem
    while ( gindex < (dim + blockDim.x) ) {
    /* FIXME PART 2 - MODIFIY PROGRAM TO USE SHARED MEMORY. */

    // Apply the stencil
        double result = 0;
        for (int offset = -RADIUS; offset <= RADIUS; offset++) {
            if ( gindex + offset < dim && gindex + offset > -1)
    	        result += in[gindex + offset];
        }
    // Store the result
        if (gindex < dim)
            out[gindex] = result;

    // Update global index and quit if we are done
        gindex += stride;

        __syncthreads();
    }
}

__global__ void stencil_shared_1D(double* in, double* out, long dim)
{
 __shared__ double array[BLOCKSIZE + 2 * RADIUS];

    long Idx = threadIdx.x + blockDim.x * blockIdx.x;
    int local_idx = threadIdx.x;
    array[RADIUS + local_idx] = in[Idx];

    __syncthreads();

    if ( ((local_idx + RADIUS) >= BLOCKSIZE) && ((Idx + RADIUS) < dim) ) {
        array[local_idx + 2* RADIUS] = in[Idx + RADIUS];
    }

    if ( ((local_idx - RADIUS) < 0) && ((Idx - RADIUS) > -1) ) {
        array[local_idx] = in[Idx - RADIUS];
    }

    __syncthreads();

    double result = 0;
    for (int offset = -RADIUS; offset <= RADIUS; offset++) {
        if (Idx + offset < dim && Idx + offset > -1)
            result += array[RADIUS + local_idx + offset];
    }
    out[Idx] = result;
}

#define True 1
#define False 0
void checkResults(double* h_in, double* h_out, int DoCheck = True)
{
    // DO NOT CHANGE THIS CODE.
    // CPU calculates the stencil from data in *h_in
    // if DoCheck is True (default) it compares it with *h_out
    // to check the operation of this code.
    // If DoCheck is set to False, it can be used to time the CPU.
    int i, j, ij, err;
    double result;
    err = 0;
    for (i = 0; i < N; i++) { // major index.
        result = 0;
        for (j = -RADIUS; j <= RADIUS; j++) {
            ij = i + j;
            if (ij >= 0 && ij < N)
                result += h_in[ij];
        }
        if (DoCheck) { // print out some errors for debugging purposes.
            if (h_out[i] != result) { // count errors.
                err++;
                if (err < 8) { // help debug
                    printf("h_out[%d]=%d should be %d\n", i, h_out[i], result);
                };
            }
        } else { // for timing purposes.
            h_out[i] = result;
        }
    }

    if (DoCheck) { // report results.
        if (err != 0) {
            printf("Error, %d elements do not match!\n", err);
        } else {
            printf("Success! All elements match CPU result.\n");
        }
    }
}

// ------------
// main program
// ------------
int main(int argc, char**argv)
{
    N = atoi(argv[1]);
    gridSize = (N-1) / BLOCKSIZE + 1;
    double *h_in, *h_out;
    double *d_in, *d_out;
    long size = N * sizeof(double);
    int i;

    // allocate host memory
    h_in = new double[N];
    h_out = new double[N];

    getDeviceProperties();

    // initialize vector
    for (i = 0; i < N; i++) {
        //    h_in[i] = i+1;
        h_in[i] = 1;
    }

    // allocate device memory
    hipMalloc((void**)&d_in, size);
    hipMalloc((void**)&d_out, size);
    cudaErrorCheck();

    // copy input data to device
    hipMemcpy(d_in, h_in, size, hipMemcpyHostToDevice);
    cudaErrorCheck();

    // Apply stencil by launching a sufficient number of blocks
    printf("\n---------------------------\n");
    printf("Launching 1D stencil kernel\n");
    printf("---------------------------\n");
    printf("Vector length     = %ld (%ld MB)\n", N,
        N * sizeof(double) / 1024 / 1024);
    printf("Stencil radius    = %d\n", RADIUS);

    //----------------------------------------------------------
    // CODE TO RUN AND TIME THE STENCIL KERNEL.
    //----------------------------------------------------------

    newline();
    printThreadSizes();
    hipEvent_t start, stop;
    start_timer(&start);
    if (atoi(argv[2])) {
        stencil_shared_1D<<<gridSize, blockSize>>>(d_in, d_out, N);
        std::cout << "stencil_shared_1D" << std::endl;
    } else {
        stencil_1D<<<gridSize, blockSize>>>(d_in, d_out, N);
        std::cout << "stencil_1D" << std::endl;
    }
    std::cout << "Elapsed time: " << stop_timer(&start, &stop) << std::endl;
    // copy results back to host
    hipMemcpy(h_out, d_out, size, hipMemcpyDeviceToHost);
    cudaErrorCheck();
    checkResults(h_in, h_out);
    //----------------------------------------------------------

    // deallocate device memory
    hipFree(d_in);
    hipFree(d_out);
    cudaErrorCheck();
    //=====================================================
    // Evaluate total time of execution with just the CPU.
    //=====================================================
    newline();
    std::cout << "Running stencil with the CPU.\n";
    start_timer(&start);
    // Use checkResults to time CPU version of the stencil with False flag.
    checkResults(h_in, h_out, False);
    std::cout << "Elapsed time: " << stop_timer(&start, &stop) << std::endl;
    //=====================================================

    // deallocate host memory
    free(h_in);
    free(h_out);

    return 0;
}
