#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>
#include <stdlib.h>
#include <ctime>
#include <cmath>
#include <limits>

using namespace std;

__global__ void sum_vectors(double *a, double *b, double *c, int size){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        c[idx] = a[idx] + b[idx];
    }
}

int check(double *a, double *b, double *c, int size) {
    for (int i = 0; i < size; ++i) {
        if (std::fabs(c[i] - (a[i] + b[i])) > std::numeric_limits<double>::epsilon()) {
            return 0;
        }
    }
    return 1;
}

int main(int argc, char **argv){
    int n = (int)strtol(argv[1], NULL, 10);

    double *h_a, *h_b, *h_c;

    size_t bytes = n * sizeof(double);

    h_a = (double *) malloc(bytes);
    h_b = (double *) malloc(bytes);
    h_c = (double *) malloc(bytes);

    for (int i = 0; i < n; i++){
        h_a[i] = sin(i) * sin(i);
        h_b[i] = cos(i) * cos(i);
    }

    double *d_a, *d_b, *d_c;

    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);
    hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);

    int blockSize, gridSize;
    blockSize = 1024;
    gridSize = (n - 1) / 1024 + 1;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    sum_vectors<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);

    hipDeviceSynchronize();
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);
    
    auto start_cpu = std::chrono::high_resolution_clock::now();

    int check_flag = check(h_a, h_b, h_c, n);

    auto stop_cpu = std::chrono::high_resolution_clock::now();
    auto elapsed_time_cpu = stop_cpu - start_cpu;

    if (check_flag) {
        std::cout << "Correct sum" << std::endl;
    } else {
        std::cout << "Not correct sum" << std::endl;
    }

    std::cout << "Cpu time: " << elapsed_time_cpu.count() / 1000 << " milliseconds" << std::endl;
    std::cout << "Gpu time: " << milliseconds << " milliseconds" << std::endl;

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    free(h_a);
    free(h_b);
    free(h_c);

    return 0;
}
