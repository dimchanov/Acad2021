
#include <stdio.h>
#include <assert.h>
#include <iostream>
#include <hip/hip_runtime.h>

#include <limits>

#ifndef MAX
#define MAX(a,b) (a > b ? a : b)
#endif

__global__ void vectorAddGPU(float *a, float *b, float *c, int N, int offset)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    
    if (idx < N)
    {
        c[offset + idx] = a[offset + idx] + b[offset + idx];
    }
}

int check(float *a, float *b, float *c, int size) {
    for (int i = 0; i < size; ++i) {
        if (std::fabs(c[i] - (a[i] + b[i])) > std::numeric_limits<double>::epsilon()) {
            return 0;
        }
    }
    return 1;
}

void sample_vec_add(int size = 1048576)
{
    int n = size;
    
    int nBytes = n*sizeof(int);
    
    float *a, *b;  // host data
    float *c;  // results
    
    a = (float *)malloc(nBytes);
    b = (float *)malloc(nBytes);
    c = (float *)malloc(nBytes);
    
    float *a_d,*b_d,*c_d;
    
    dim3 block(256);
    dim3 grid((unsigned int)ceil(n/(float)block.x));
    
    for(int i=0;i<n;i++)
    {
        a[i] = rand() / (float)RAND_MAX;
        b[i] = rand() / (float)RAND_MAX;
        c[i] = 0;
    }
    
    printf("Allocating device memory on host..\n");
    
    hipMalloc((void **)&a_d,n*sizeof(float));
    hipMalloc((void **)&b_d,n*sizeof(float));
    hipMalloc((void **)&c_d,n*sizeof(float));
    
    printf("Copying to device..\n");
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipEventRecord(start);
    
    hipMemcpy(a_d,a,n*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(b_d,b,n*sizeof(float), hipMemcpyHostToDevice);
    
    printf("Doing GPU Vector add\n");
    
    vectorAddGPU<<<grid, block>>>(a_d, b_d, c_d, n, 0);
    hipMemcpy(c,c_d,n*sizeof(float), hipMemcpyDeviceToHost);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("GPU time: %f ms\n", milliseconds);
    
    hipDeviceSynchronize();

    int res = check(a, b, c, n);
    if (res) {
        std::cout << "Correct result" << std::endl;
    } else {
        std::cout << "Not correct result" << std::endl;
    }
    
    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);
}

void streams_vec_add(int n_streams, int size = 1048576)
{
    int n = size;
    
    float *a, *b;  // host data
    float *c;  // results
    
    hipHostAlloc( (void**) &a, n * sizeof(float) ,hipHostMallocDefault );
    hipHostAlloc( (void**) &b, n * sizeof(float) ,hipHostMallocDefault );
    hipHostAlloc( (void**) &c, n * sizeof(float) ,hipHostMallocDefault );
    
    float *a_d,*b_d,*c_d;
    
    for(int i=0;i<n;i++)
    {
        a[i] = rand() / (float)RAND_MAX;
        b[i] = rand() / (float)RAND_MAX;
        c[i] = 0;
    }
    
    printf("Allocating device memory on host..\n");

    hipMalloc((void **)&a_d,n*sizeof(float));
    hipMalloc((void **)&b_d,n*sizeof(float));
    hipMalloc((void **)&c_d,n*sizeof(float));
    
    printf("Copying to device..\n");
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipEventRecord(start);
    
    printf("Doing GPU-stream Vector add\n");
    
    
    const int NbStreams = n_streams;
    const int StreamSize = n / NbStreams;
    hipStream_t Stream[NbStreams];
    for ( int i = 0; i < NbStreams; i++ )
        hipStreamCreate(&Stream[i]);
    

    for ( int i = 0; i < NbStreams; i++ )
    {
        int Offset = i * StreamSize;
        
        hipMemcpyAsync(&a_d[Offset], &a[Offset], StreamSize * sizeof(float), hipMemcpyHostToDevice, Stream[ i ]);
        hipMemcpyAsync(&b_d[Offset], &b[Offset], StreamSize * sizeof(float), hipMemcpyHostToDevice, Stream[ i ]);
        hipMemcpyAsync(&c_d[Offset], &c[Offset], StreamSize * sizeof(float), hipMemcpyHostToDevice, Stream[ i ]);
        
        dim3 block(1024);
        dim3 grid((StreamSize - 1)/1024 + 1);
        vectorAddGPU<<<grid, block, 0, Stream[i]>>>(a_d, b_d, c_d, StreamSize, Offset);
    
        hipMemcpyAsync(&a[Offset], &a_d[Offset], StreamSize * sizeof(float), hipMemcpyDeviceToHost, Stream[ i ]);
        hipMemcpyAsync(&b[Offset], &b_d[Offset], StreamSize * sizeof(float), hipMemcpyDeviceToHost, Stream[ i ]);
        hipMemcpyAsync(&c[Offset], &c_d[Offset], StreamSize * sizeof(float), hipMemcpyDeviceToHost, Stream[ i ]);

    }
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "STREAMS NUMBERS: " << NbStreams << std::endl;
    printf("GPU-stream time: %f ms\n", milliseconds);
    
    hipDeviceSynchronize();
    int res = check(a, b, c, n);
    if (res) {
        std::cout << "Correct result" << std::endl;
    } else {
        std::cout << "Not correct result" << std::endl;
    }
    
    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);
    
    hipHostFree(a);
    hipHostFree(b);
    hipHostFree(c);
}


int main(int argc, char **argv)
{
    sample_vec_add(atoi(argv[1]));
    sample_vec_add(atoi(argv[1]));
    int n_streams = (argc == 3) ? atoi(argv[2]) : 8;
    std::cout << "=================================================" << std::endl;
    std::cout << "STREAMS NUMBERS: " << n_streams << std::endl;
    streams_vec_add(n_streams, atoi(argv[1]));

    return 0;
}