#include "hip/hip_runtime.h"
//
//  main.cpp
//  
//
//  Created by Elijah Afanasiev on 25.09.2018.
//
//

// System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#ifndef MAX
#define MAX(a,b) (a > b ? a : b)
#endif

__global__ void vectorAddGPU(float *a, float *b, float *c, int N)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    
    if (idx < N)
    {
        c[idx] = a[idx] + b[idx];
    }
}

void unified_samle(int size = 1048576)
{
    int n = size;
    int bytes = size * sizeof(float);

    float *h_a, *h_b, *h_c;
    
    dim3 block(256);
    dim3 grid((unsigned int)ceil(n/(float)block.x));

    printf("Allocating device unified memory on host and device\n");
    hipMallocManaged(&h_a, bytes);
    hipMallocManaged(&h_b, bytes);
    hipMallocManaged(&h_c, bytes);

    for(int i=0;i<n;i++)
    {
        h_a[i] = rand() / (float)RAND_MAX;
        h_b[i] = rand() / (float)RAND_MAX;
    }
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipEventRecord(start);
    
    printf("Doing GPU Vector add\n");
    
    vectorAddGPU<<<grid, block>>>(h_a, h_b, h_c, n);
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Unified memory time: %f ms\n", milliseconds);

    hipDeviceSynchronize();
}

void pinned_samle(int size = 1048576)
{
    int n = size;
    int bytes = size * sizeof(float);

    float *h_a, *h_b, *h_c;
    float *d_a, *d_b, *d_c;
    
    dim3 block(256);
    dim3 grid((unsigned int)ceil(n/(float)block.x));

    printf("Allocating device pinned memory on host..\n");
    hipHostMalloc(&h_a, bytes);
    hipHostMalloc(&h_b, bytes);
    hipHostMalloc(&h_c, bytes);
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);

    for(int i=0;i<n;i++)
    {
        h_a[i] = rand() / (float)RAND_MAX;
        h_b[i] = rand() / (float)RAND_MAX;
        h_c[i] = 0;
    }
    printf("Copying to device..\n");
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipEventRecord(start);
    
    hipMemcpy(d_a, h_a, n*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, n*sizeof(float), hipMemcpyHostToDevice);
    
    printf("Doing GPU Vector add\n");
    
    vectorAddGPU<<<grid, block>>>(d_a, d_b, d_c, n);
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Pinned memory time: %f ms\n", milliseconds);

    hipDeviceSynchronize();
    
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

}

void usual_sample(int size = 1048576)
{
    int n = size;
    
    int nBytes = n*sizeof(float);
    
    float *a, *b;  // host data
    float *c;  // results
    
    a = (float *)malloc(nBytes);
    b = (float *)malloc(nBytes);
    c = (float *)malloc(nBytes);
    
    float *a_d,*b_d,*c_d;
    
    dim3 block(256);
    dim3 grid((unsigned int)ceil(n/(float)block.x));
    
    for(int i=0;i<n;i++)
    {
        a[i] = rand() / (float)RAND_MAX;
        b[i] = rand() / (float)RAND_MAX;
        c[i] = 0;
    }
    
    printf("Allocating device memory on host..\n");
    
    hipMalloc((void **)&a_d,n*sizeof(float));
    hipMalloc((void **)&b_d,n*sizeof(float));
    hipMalloc((void **)&c_d,n*sizeof(float));
    
    printf("Copying to device..\n");
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipEventRecord(start);
    
    hipMemcpy(a_d,a,n*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(b_d,b,n*sizeof(float), hipMemcpyHostToDevice);
    
    printf("Doing GPU Vector add\n");
    
    vectorAddGPU<<<grid, block>>>(a_d, b_d, c_d, n);
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("time: %f ms\n", milliseconds);
    
    hipDeviceSynchronize();
    
    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);
}


int main(int argc, char **argv)
{
    usual_sample(atoi(argv[1]));
    pinned_samle(atoi(argv[1]));
    unified_samle(atoi(argv[1]));
    
    return 0;
}
