#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>
#include <stdlib.h>
#include <ctime>
#include <cmath>
#include <limits>

using namespace std;

__global__ void sum_vectors(double *a, double *b, double *c, int size){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        c[idx] = a[idx] + b[idx];
    }
}

int check(double *a, double *b, double *c, int size) {
    for (int i = 0; i < size; ++i) {
        if (std::fabs(c[i] - (a[i] + b[i])) > std::numeric_limits<double>::epsilon()) {
            return 0;
        }
    }
    return 1;
}

int main(int argc, char **argv){
    int n = (int)strtol(argv[1], NULL, 10);

    double *h_a, *h_b, *h_c;

    size_t bytes = n * sizeof(double);

    hipHostMalloc(&h_a, bytes, hipHostMallocDefault);
    hipHostMalloc(&h_b, bytes, hipHostMallocDefault);
    hipHostMalloc(&h_c, bytes, hipHostMallocDefault);

    for (int i = 0; i < n; i++){
        h_a[i] = i;
        h_b[i] = 3 * i;
    }

    double *d_a, *d_b, *d_c;
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);

        
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);

    int blockSize, gridSize;
    blockSize = 1024;
    gridSize = (n - 1) / 1024 + 1;


    sum_vectors<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);


    hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    
    auto start_cpu = std::chrono::high_resolution_clock::now();

    int check_flag = check(h_a, h_b, h_c, n);

    auto stop_cpu = std::chrono::high_resolution_clock::now();
    auto elapsed_time_cpu = stop_cpu - start_cpu;

    if (check_flag) {
        std::cout << "Correct sum" << std::endl;
    } else {
        std::cout << "Not correct sum" << std::endl;
    }

    std::cout << "Cpu time: " << elapsed_time_cpu.count() / 1000 << " milliseconds" << std::endl;
    std::cout << "Gpu time: " << milliseconds << " milliseconds" << std::endl;

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    return 0;
}
