#include <iostream>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <ctime>
#include <cmath>
#include <limits>

__global__ void sum_vectors(double *a, double *b, double *c, int size){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        c[idx] = a[idx] + b[idx];
    }
}

int check(double *a, double *b, double *c, int size) {
    for (int i = 0; i < size; ++i) {
        if (std::fabs(c[i] - (a[i] + b[i])) > std::numeric_limits<double>::epsilon()) {
            return 0;
        }
    }
    return 1;
}

int main(int argc, char **argv){
    int n = atoi(argv[1]);
    int n_device = (n - 1)/2 + 1;
    int bytes_device = n_device * sizeof(double);

    double *h_a, *h_b, *h_c;
    size_t bytes = n * sizeof(double);

    h_a = (double *) malloc(bytes);
    h_b = (double *) malloc(bytes);
    h_c = (double *) malloc(bytes);
    hipHostRegister(h_a, bytes, 0);
    hipHostRegister(h_b, bytes, 0);
    hipHostRegister(h_c, bytes, 0);

    for (int i = 0; i < n; i++){
        h_a[i] = i;
        h_b[i] = 3 * i;
    }

    double *d_a1, *d_b1, *d_c1;
    double *d_a2, *d_b2, *d_c2;
    hipMalloc(&d_a1, bytes_device);
    hipMalloc(&d_b1, bytes_device);
    hipMalloc(&d_c1, bytes_device);
    hipSetDevice(1);
    hipMalloc(&d_a2, bytes_device);
    hipMalloc(&d_b2, bytes_device);
    hipMalloc(&d_c2, bytes_device);
    hipSetDevice(0);
    

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    int blockSize, gridSize;
    blockSize = 1024;
    gridSize = (n_device - 1) / 1024 + 1;

    hipSetDevice(0);
    hipMemcpyAsync(d_a1, &h_a[0], bytes_device, hipMemcpyHostToDevice);
    hipMemcpyAsync(d_b1, &h_b[0], bytes_device, hipMemcpyHostToDevice);

    sum_vectors<<<gridSize, blockSize>>>(d_a1, d_b1, d_c1, n_device);

    hipMemcpyAsync(&h_c[0], d_c1, bytes_device, hipMemcpyDeviceToHost);

    hipSetDevice(1);
    hipMemcpyAsync(d_a2, &h_a[n_device], bytes_device, hipMemcpyHostToDevice);
    hipMemcpyAsync(d_b2, &h_b[n_device], bytes_device, hipMemcpyHostToDevice);

    sum_vectors<<<gridSize, blockSize>>>(d_a2, d_b2, d_c2, n_device);

    hipMemcpyAsync(&h_c[n_device], d_c2, bytes_device, hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
    hipSetDevice(0);
    hipDeviceSynchronize();

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Gpu time: " << milliseconds << " milliseconds" << std::endl;
    
    int res = check(h_a, h_b, h_c, n);

    if (res) {
        std::cout << "Correct result" << std::endl;
    } else {
        std::cout << "Not correct result" << std::endl;
    }

    hipFree(d_a1);
    hipFree(d_b1);
    hipFree(d_c1);
    hipFree(d_a2);
    hipFree(d_b2);
    hipFree(d_c2);
    hipHostUnregister(h_a);
    hipHostUnregister(h_b);
    hipHostUnregister(h_c);
    free(h_a);
    free(h_b);
    free(h_c);

    return 0;
}
